#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
#include<hipsparse.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>

typedef float real;
#define MAX_STRING 100
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define bool int
#define false 0
#define true 1
#define max(a,b)(a>b)?a:b
const real zero = 0.0;
const real one = 1.0;
const real negOne = -1.0;

hipsparseHandle_t handle_sparse = 0;
hipsparseMatDescr_t descr_sparse = 0;
hipblasHandle_t handle_blas = 0;
hipError_t cudaStat;       //for cuda errors
dim3 threadsPerBlock(16);

/* define variables */
char filename[100];         //the file of V, store as sparse matrix
int gpuid = 0;              //GPU to use
int m, n = 2, k;              //V=WH, V:m*k, W:m*n, H:n*k
real tol = 0.001;
real timelimit = 100;
int maxiter = 10;
int lineNumber = -1;        //line number(positive value) of V
FILE *file;                 //file handle
char _str[MAX_STRING];      //a black hole for string reading
int tmpRow, tmpCol;         //tmp variables for reading sparse matrix index
real tmpVal;                //tmp variable for reading sparse matrix value

int *VRowIndexHost = 0;     //row index of V in host
int *VColIndexHost = 0;     //column index of V in host
real *VHost = 0;            //value of V in host
real *WHost = 0;            //value of W in host
real *HHost = 0;            //value of H in host

int *VRowCoo = 0;           //row index of V in GPU in COO format, for reading data only
int *VRow = 0;              //row index of V in GPU
int *VCol = 0;              //col index of V in GPU
real *V = 0;                //V in GPU
real *W = 0, *H = 0;        //W,H in GPU

/* a macro for free memory*/
#define CLEANUP(s)                                  \
do {                                                \
    printf ("%s\n", s);                             \
    if (WHost) free(WHost);                         \
    if (HHost) free(HHost);                         \
    if (VRow) hipFree(VRow);                       \
    if (VCol) hipFree(VCol);                       \
    if (V) hipFree(V);                             \
    if (W) hipFree(W);                             \
    if (H) hipFree(H);                             \
    hipsparseDestroy(handle_sparse);                 \
    hipsparseDestroyMatDescr(descr_sparse);          \
    hipblasDestroy(handle_blas);                     \
    hipDeviceReset();                              \
    fflush (stdout);                                \
} while (0)

void randomInit(real *data, int p){
    int i = 0;
    for (; i < p; ++i)
        data[i] = rand() / (real)RAND_MAX;
}

/* print a matrix of size row*col */
void outPutMatrix(int row, int col, real *A){
    int i, j;
    for(i = 0; i < row; i++){
        for(j = 0; j < col; j++)
            printf("%10.4f ", A[IDX2C(i,j,row)]);
        printf("\n");
    }
}

//clip negative value
__global__ void clipNegative(real *A, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N && A[i] < 0)
        A[i] = 0;
}

//projgrad = norm(grad(grad < 0 | H >0)), let tmpvec is the useful values
__global__ void getUsefulGrad(real *grad, real *H, real *tmpvec, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        if(grad[i] < 0 || H[i] < 0)
            tmpvec[i] = grad[i];
}

//projnorm = norm([gradW(gradW<0 | W>0); gradH(gradH<0 | H>0)]);
__global__ void getUsefulGrad2(real *grad, real *H, real *tmpvec, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        if(grad[i] < 0 || H[i] > 0)
            tmpvec[i] = grad[i];
}

void initVaribles(){
    srand((unsigned)time(NULL));

    /* allocate memory and initial */
    if((file = fopen(filename, "r")) == NULL){
        printf("File %s not found!\n", filename);
        exit(1);
    }
    while(fgets(_str, sizeof(_str), file))
        lineNumber++;
    fclose(file);
    printf("Line number(positive value) of V: %d\n", lineNumber);
    VRowIndexHost = (int *)malloc(lineNumber*sizeof(int));
    VColIndexHost = (int *)malloc(lineNumber*sizeof(int));
    VHost = (real *)malloc(lineNumber*sizeof(real));

    file = fopen(filename, "r");
    fscanf(file, "%d %d", &m, &k);
    printf("Matrix shape of m n k: %d %d %d\n", m, n, k);

    WHost = (real *)malloc(m*n*sizeof(real));
    randomInit(WHost, m*n);
    HHost = (real *)malloc(n*k*sizeof(real));
    randomInit(HHost, n*k);

    int i = 0;
    while(fscanf(file, "%d %d %f", &tmpRow, &tmpCol, &tmpVal) != EOF){
        *(VRowIndexHost+i) = tmpRow;
        *(VColIndexHost+i) = tmpCol;
        *(VHost+i) = tmpVal;
        i++;
    }
    fclose(file);

    cudaStat = hipSetDevice(gpuid);
    if(cudaStat != hipSuccess){
        CLEANUP("Device not found, check your gpuid!");
        exit(1);
    }
    /* setup cusparse and cublas library */
    hipsparseCreate(&handle_sparse);
    hipsparseCreateMatDescr(&descr_sparse);
    hipsparseSetMatType(descr_sparse,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr_sparse,HIPSPARSE_INDEX_BASE_ZERO);
    hipblasCreate(&handle_blas);
}

/* shipping data to GPU */
void shipping(){
    hipMalloc((void**)&VRowCoo, lineNumber*sizeof(int));
    hipMalloc((void**)&VCol, lineNumber*sizeof(int));
    hipMalloc((void**)&V, lineNumber*sizeof(real));
    hipMalloc((void**)&W, m*n*sizeof(real));
    hipMalloc((void**)&H, n*k*sizeof(real));

    hipMemcpy(VRowCoo, VRowIndexHost, (size_t)(lineNumber*sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(VCol, VColIndexHost, (size_t)(lineNumber*sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(V, VHost, (size_t)(lineNumber*sizeof(real)), hipMemcpyHostToDevice);
    hipMemcpy(W, WHost, (size_t)(m*n*sizeof(real)), hipMemcpyHostToDevice);
    hipMemcpy(H, HHost, (size_t)(n*k*sizeof(real)), hipMemcpyHostToDevice);

    /* convert V from COO 2 CSR format */
    hipMalloc((void**)&VRow,(m+1)*sizeof(int));
    hipsparseXcoo2csr(handle_sparse, VRowCoo, lineNumber, m, VRow, HIPSPARSE_INDEX_BASE_ZERO);

    //print V for test
    real *Vdense, *VdenseHost;
    hipMalloc((void**)&Vdense, m*k*sizeof(real));
    hipsparseScsr2dense(handle_sparse, m, k, descr_sparse, V, VRow, VCol, Vdense, m);
    VdenseHost = (real *)malloc(m*k*sizeof(real));
    hipMemcpy(VdenseHost, Vdense, (size_t)(m*k*sizeof(real)), hipMemcpyDeviceToHost);
    printf("V:\n");
    outPutMatrix(m, k, VdenseHost);

    /* free some useless variables */
    if (VHost) free(VHost);
    if (VRowIndexHost) free(VRowIndexHost);
    if (VColIndexHost) free(VColIndexHost);
    if (VRowCoo) hipFree(VRowCoo);
}

void subprob(real *V, hipsparseOperation_t transV, int rowV, int colV, real *W, real *Hinit,
            int mm, int nn, int kk, real tol, int maxiter, real *H, real *grad, int *ite){
    //H = Hinit
    hipMemcpy(H, Hinit, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);

    //WtV = W'*V;
    real *WtV = 0, *VtW = 0;
    hipMalloc((void**)&WtV, nn*kk*sizeof(real));
    hipMalloc((void**)&VtW, kk*nn*sizeof(real));
    hipsparseScsrmm(handle_sparse, transV, rowV, nn, colV, lineNumber, &one, descr_sparse, V, VRow, VCol, W, mm, &zero, VtW, kk);//VtW = V'*W
    hipblasSgeam(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, nn, kk, &one, VtW, kk, &zero, WtV, nn, WtV, nn);//WtV = (VtW)'
    hipFree(VtW);

    //WtW = W'*W;
    real *WtW = 0;
    hipMalloc((void**)&WtW, m*m*sizeof(real));
    hipblasSgemm(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, nn, nn, mm, &one, W, mm, W, mm, &zero, WtW, nn);

    real alpha = 1;
    real beta = 0.1;

    real *Hn = 0;
    hipMalloc((void**)&Hn, nn*kk*sizeof(real));
    hipMemcpy(Hn, H, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);

    real *d = 0;
    hipMalloc((void**)&d, nn*kk*sizeof(real));
    hipMemset(d, 0, nn*kk*sizeof(real));

    real *WtWd = 0;
    hipMalloc((void**)&WtWd, nn*kk*sizeof(real));
    hipMemset(WtWd, 0, nn*kk*sizeof(real));

    real *Hp = 0;
    hipMalloc((void**)&Hp, nn*kk*sizeof(real));
    hipMemset(Hp, 0, nn*kk*sizeof(real));

    real *Hnpp = 0;
    hipMalloc((void**)&Hnpp, nn*kk*sizeof(real));
    hipMemset(Hnpp, 0, nn*kk*sizeof(real));

    real *tmpvec = 0;
    hipMalloc((void**)&tmpvec, nn*kk*sizeof(real));

    int iter = 0;
    for(iter = 1; iter <= maxiter; iter++){
        //grad = WtW*H - WtV;
        hipMemcpy(grad, WtV, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);//grad = WtV (tmp step)
        hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, nn, kk, nn, &one, WtW, nn, H, nn, &negOne, grad, nn);//grad = WtW*H - WtV;
        //projgrad = norm(grad(grad < 0 | H >0))
        hipMemset(tmpvec, 0, nn*kk*sizeof(real));
        dim3 num1(nn*kk / threadsPerBlock.x + 1);
        getUsefulGrad<<<num1, threadsPerBlock>>>(grad, H, tmpvec, nn*kk);
        real projgrad = 0;
        hipblasSnrm2(handle_blas, nn*kk, tmpvec, 1, &projgrad);
        //printf("projgrad %f\n", projgrad);
        if (projgrad < tol)
            break;

        int inner_iter = 1;
        for(; inner_iter <= 20; inner_iter++){
            //Hn = max(H - alpha*grad, 0); d = Hn-H;
            hipMemcpy(Hn, H, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
            real nalpha = -alpha;
            hipblasSaxpy(handle_blas, nn*kk, &nalpha, grad, 1, Hn, 1);
            dim3 num2(nn*kk / threadsPerBlock.x + 1);
            clipNegative<<<num2, threadsPerBlock>>>(Hn, nn*kk);
            hipMemcpy(d, Hn, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
            hipblasSaxpy(handle_blas, nn*kk, &negOne, H, 1, d, 1);

            //gradd=sum(sum(grad.*d)); dQd = sum(sum((WtW*d).*d));
            real gradd = 0, dQd = 0;
            hipblasSdot(handle_blas, nn*kk, grad, 1, d, 1, &gradd);
            hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, nn, kk, nn ,&one, WtW, nn, d, nn, &zero, WtWd, nn);
            hipblasSdot(handle_blas, nn*kk, WtWd, 1, d, 1, &dQd);
            bool suff_decr = 0.99*gradd + 0.5*dQd < 0;
            bool decr_alpha = true;
            if (inner_iter == 1){
                decr_alpha = ~suff_decr;
                hipMemcpy(Hp, H, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
            }
            if(decr_alpha){
                if(suff_decr){
                    hipMemcpy(H, Hn, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
                    break;
                }
                else
                    alpha = alpha * beta;
            }
            else{
                hipMemcpy(Hnpp, Hn, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
                hipblasSaxpy(handle_blas, nn*kk, &negOne, Hp, 1, Hnpp, 1);
                real test = 0;
                hipblasSnrm2(handle_blas, nn*kk, Hnpp, 1, &test);
                if(~suff_decr || test == 0){
                    hipMemcpy(H, Hp, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
                    break;
                }
                else{
                    alpha = alpha/beta;
                    hipMemcpy(Hp, Hn, nn*kk*sizeof(real), hipMemcpyDeviceToDevice);
                }
            }
        }
    }
    *ite = iter;
}

void NMF(){
    clock_t initt = time(NULL);

    //gradW = W*(H*H') - V*H';
    real *HHt = 0, *gradW = 0, *gradWt = 0,*VHt = 0;
    hipMalloc((void**)&HHt, n*n*sizeof(real));
    hipMalloc((void**)&gradW, m*n*sizeof(real));
    hipMalloc((void**)&gradWt, m*n*sizeof(real));
    hipMalloc((void**)&VHt, m*n*sizeof(real));
    hipsparseScsrmm2(handle_sparse, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            m, n, k,lineNumber, &one, descr_sparse, V, VRow, VCol, H, n, &zero, VHt, m); //VHt = V*H'
    hipMemcpy(gradW, VHt, m*n*sizeof(real), hipMemcpyDeviceToDevice); //gradW = VHt (tmp step)
    hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k, &one, H, n, H, n, &zero, HHt, n); //HHt = H*H'
    hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, n, &one, W, m, HHt, n, &negOne, gradW, m); //gradW = W*(H*H') - V*H'

    //gradH = (W'*W)*H - W'*V;
    real *WtW = 0, *gradH = 0, *WtV = 0, *VtW = 0;
    hipMalloc((void**)&WtW, m*m*sizeof(real));
    hipMalloc((void**)&gradH, n*k*sizeof(real));
    hipMalloc((void**)&WtV, n*k*sizeof(real));
    hipMalloc((void**)&VtW, k*n*sizeof(real));
    hipsparseScsrmm(handle_sparse, HIPSPARSE_OPERATION_TRANSPOSE, k, n, m,
            lineNumber, &one, descr_sparse, V, VRow, VCol, W, m, &zero, VtW, k);//VtW = V'*W
    hipblasSgeam(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, n, k, &one, VtW, k, &zero, WtV, n, WtV, n);//WtV = (VtW)'
    hipblasSgemm(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &one, W, m, W, m, &zero, WtW, n);//WtW = W'*W
    hipMemcpy(gradH, WtV, n*k*sizeof(real), hipMemcpyDeviceToDevice);//gradH = WtV (tmp step)
    hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, k, n, &one, WtW, n, H, n, &negOne, gradH, n);//gradH = WtW*H - WtV;

    //initgrad = norm(gradW) + norm(gradH);
    real initgrad = 0, tmpgrad = 0;
    hipblasSdot(handle_blas, m*n, gradW, 1, gradW, 1, &initgrad);
    hipblasSdot(handle_blas, n*k, gradH, 1, gradH, 1, &tmpgrad);
    initgrad += tmpgrad;
    initgrad = sqrt(initgrad);
    printf("Init gradient norm %f\n", initgrad);
    real tolW = initgrad*max(0.001,tol);
    real tolH = tolW;

    real *tmpvec, *tmpvec2; //W, H
    hipMalloc((void**)&tmpvec, m*n*sizeof(real));
    hipMemset(tmpvec, 0, m*n*sizeof(real));
    hipMalloc((void**)&tmpvec2, n*k*sizeof(real));
    hipMemset(tmpvec2, 0, n*k*sizeof(real));

    real *Wt, *Ht; //Wt, Ht
    hipMalloc((void**)&Wt, m*n*sizeof(real));
    hipMalloc((void**)&Ht, n*k*sizeof(real));

    int iter = 0;
    real projnorm = 0, tmpnorm = 0, lastnorm = 0;
    for(iter = 1; iter <= maxiter; iter++){
        //stopping condition
        //projnorm = norm([gradW(gradW<0 | W>0); gradH(gradH<0 | H>0)]);
        dim3 num3(m*n / threadsPerBlock.x + 1);
        getUsefulGrad2<<<num3, threadsPerBlock>>>(gradW, W, tmpvec, m*n);
        hipblasSdot(handle_blas, m*n, tmpvec, 1, tmpvec, 1, &projnorm);
        dim3 num4(n*k / threadsPerBlock.x + 1);
        getUsefulGrad2<<<num4, threadsPerBlock>>>(gradH, H, tmpvec2, n*k);
        hipblasSdot(handle_blas, n*k, tmpvec2, 1, tmpvec2, 1, &tmpnorm);
        projnorm += tmpnorm;
        projnorm = sqrt(projnorm);
        printf("Iter %d, projnorm %f\n", iter, projnorm);
        if(iter != 1 && projnorm == lastnorm)
            break;
        if(projnorm < tol*initgrad || time(NULL)-initt > timelimit)
            break;
        lastnorm = projnorm;

        //update W, Vt = HtWt, then Wt is the same as H before
        hipblasSgeam(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &one, W, m, &zero, Wt, n, Wt, n); //Wt
        hipblasSgeam(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, k, n, &one, H, n, &zero, Ht, k, Ht, k); //Ht
        int iterW = 0;
        subprob(V, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, k, Ht, Wt, k, n, m, tolW, 1000, W, gradW, &iterW);
        if(iterW == 1 && tolW > 0.000001)
            tolW = 0.1 * tolW;
        //W = W' , gradW = gradW'
        hipMemcpy(Wt, W, (size_t)(m*n*sizeof(real)), hipMemcpyDeviceToDevice);
        hipMemcpy(gradWt, gradW, (size_t)(m*n*sizeof(real)), hipMemcpyDeviceToDevice);
        hipblasSgeam(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &one, Wt, n, &zero, W, m, W, m);
        hipblasSgeam(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &one, gradWt, n, &zero, gradW, m, gradW, m);

        //update H
        int iterH = 0;
        subprob(V, HIPSPARSE_OPERATION_TRANSPOSE, m, k, W, H, m, n, k, tolH, 1000, H, gradH, &iterH);
        if(iterH == 1 && tolH > 0.000001)
            tolH = 0.1 * tolH;
        //printf("HH, %d\n", iterH);
    }
}

/* shipping back to host */
void backHost(){
    hipMemcpy(WHost, W, (size_t)(m*n*sizeof(real)), hipMemcpyDeviceToHost);
    hipMemcpy(HHost, H, (size_t)(n*k*sizeof(real)), hipMemcpyDeviceToHost);
}

int ArgPos(char *str, int argc, char **argv){
    int a;
    for (a = 1; a < argc; a++)
        if (!strcmp(str, argv[a])){
            if (a == argc - 1){
                printf("Argument missing for %s\n", str);
                exit(1);
            }
        return a;
    }
    return -1;
}

int main(int argc, char **argv){
    int i, j;
    if(argc == 1){
        printf("NMF: Non-negative Matrix Factorization\n\n");
        printf("Options:\n");
        printf("Parameters for training:\n");
        printf("\t-train <file>\n");
        printf("\t\tUse data from <file> to train the model;\n");
        printf("\t-factor <int>\n");
        printf("\t\tfactor number; default is 3\n");
        printf("\t-maxiter <int>\n");
        printf("\t\tmaxiter for main loop; default is 10\n");
        printf("\t-timelimit <int>\n");
        printf("\t\ttimelimit for training; default is 100s\n");
        printf("\t-gpuid <int>\n");
        printf("\t\twhich gpu to use; default is 0\n");
        printf("\nExamples:\n");
        printf("./NMF_gd -train test.txt -factor 3 -maxiter 10 -timelimit 100 -gpuid 0\n\n");
        return 0;
    }
    if ((i = ArgPos((char *)"-train", argc, argv)) > 0) strcpy(filename, argv[i + 1]);
    if ((i = ArgPos((char *)"-factor", argc, argv)) > 0) n = atoi(argv[i + 1]);
    if ((i = ArgPos((char *)"-maxiter", argc, argv)) > 0) maxiter = atoi(argv[i + 1]);
    if ((i = ArgPos((char *)"-timelimit", argc, argv)) > 0) timelimit = atoi(argv[i + 1]);
    if ((i = ArgPos((char *)"-gpuid", argc, argv)) > 0) gpuid = atoi(argv[i + 1]);

    initVaribles();
    shipping();
    NMF();

    //save result
    backHost();
    FILE *f = fopen("W.txt", "w");
    for(i = 0; i < m; i++){
        for(j = 0; j < n; j++)
            fprintf(f, "%.4f ", *(WHost + IDX2C(i,j,m)));
        fprintf(f, "\n");
    }
    fclose(f);
    f = fopen("H.txt", "w");
    for(i = 0; i < n; i++){
        for(j = 0; j < k; j++)
            fprintf(f, "%.4f ", *(HHost + IDX2C(i,j,n)));
        fprintf(f, "\n");
    }
    fclose(f);

    /*
    printf("Result:\n");
    printf("W:\n");
    outPutMatrix(m, n, WHost);
    printf("H:\n");
    outPutMatrix(n, k, HHost);
    */

    //slowTest
    /*
    real *Vdense, *VdenseHost=0;
    hipMalloc((void**)&Vdense, m*k*sizeof(real));
    VdenseHost = (real *)malloc(m*k*sizeof(*VdenseHost));
    hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &one, W, m, H, n, &zero, Vdense, m);
    hipMemcpy(VdenseHost, Vdense, (size_t)(m*k*sizeof(real)), hipMemcpyDeviceToHost);
    printf("WH:\n");
    outPutMatrix(m, k, VdenseHost);
    */

    CLEANUP("end.");
    return 0;
}
